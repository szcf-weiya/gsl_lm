#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <gsl/gsl_matrix.h>
#include <gsl/gsl_vector.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cuLUsolve.h"
#include "cuMultifit.h"

int cuMultifit(const double *X, int n, int p, const double *Y, double *coef, double *cov)
{
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
  hipError_t cudaStat1 = hipSuccess;
  hipError_t cudaStat2 = hipSuccess;
  hipError_t cudaStat3 = hipSuccess;


  const int lda = n;
  double *C;
  C = (double*)malloc(sizeof(double)*p*p);

  double *d_X = NULL;
  double *d_C = NULL;
  double *d_Y = NULL;

  // create cublas handle
  cublas_status = hipblasCreate(&cublasH);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

  // copy to device
  cudaStat1 = hipMalloc ((void**)&d_X, sizeof(double) * lda * p);
  cudaStat2 = hipMalloc ((void**)&d_C, sizeof(double) * p * p);
  cudaStat3 = hipMalloc ((void**)&d_Y, sizeof(double) * p);
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3); //check!!

  cudaStat1 = hipMemcpy(d_X, X, sizeof(double) * lda * p, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat1);
  cudaStat2 = hipMemcpy(d_Y, Y, sizeof(double) * p, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat2);
  double alpha_v = 1.0;
  double beta_v = 0.0;
  const double *alpha = &alpha_v, *beta = &beta_v; //check!!
  printf("hello\n");
  printf("%f\n", *alpha);
  // d_C = d_X^T d_X
  cublas_status = hipblasDgemm(cublasH,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           p, n, p,
                           alpha,
                           d_X, n,
                           d_X, n,
                           beta,
                           d_C, p);
  cudaStat1 = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat1);
  printf("finish X'X\n");
  // copy d_C to C
  cudaStat1 = hipMemcpy(C, d_C, sizeof(double)*p*p, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);

  // inv(C)
  gsl_matrix *B = gsl_matrix_alloc(p, p);
  gsl_matrix_set_identity(B);
  cuda_LU_solve(C, p, B->data, p);
  cudaStat1 = hipMemcpy(d_C, B->data, sizeof(double)*p*p, hipMemcpyHostToDevice);
  gsl_matrix_free(B);
  assert(hipSuccess == cudaStat1);
  printf("finish inv(C)\n");
  // d_Y = d_X^T * d_Y
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           alpha,
                           d_X, n,
                           d_Y, 1,
                           beta,
                           d_Y, 1);
  cudaStat1 = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat1);

  // inv(C) * d_Y
  // due to by-column in gpu while by-row in gsl, C need to be transpose
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           p, p,
                           alpha,
                           d_C, p,
                           d_Y, 1,
                           beta,
                           d_Y, 1);
  cudaStat1 = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat1);

  // copy to coef
  cudaStat1 = hipMemcpy(d_Y, coef, sizeof(double) * p, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);

  hipFree(d_X);
  hipFree(d_Y);
  hipFree(d_C);

  hipblasDestroy(cublasH);
  hipDeviceReset();
  return 0;
}
