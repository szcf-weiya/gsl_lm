#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <gsl/gsl_matrix.h>
#include <gsl/gsl_vector.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cuLUsolve.h"
#include "cuMultifit.h"

int cuMultifit(const double *X, int n, int p, const double *Y, double *coef, double *cov)
{
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
  hipError_t cudaStat1 = hipSuccess;
  hipError_t cudaStat2 = hipSuccess;
  hipError_t cudaStat3 = hipSuccess;
  hipError_t cudaStat4 = hipSuccess;
  hipError_t cudaStat5 = hipSuccess;



  const int lda = n;
  double *C;
  C = (double*)malloc(sizeof(double)*p*p);

  double *d_X = NULL;
  double *d_C = NULL;
  double *d_Y = NULL;
  double *d_coef = NULL;
  double *d_coef2 = NULL;


  // create cublas handle
  cublas_status = hipblasCreate(&cublasH);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

  // copy to device
  cudaStat1 = hipMalloc ((void**)&d_X, sizeof(double) * lda * p);
  cudaStat2 = hipMalloc ((void**)&d_C, sizeof(double) * p * p);
  cudaStat3 = hipMalloc ((void**)&d_Y, sizeof(double) * n);
  cudaStat4 = hipMalloc ((void**)&d_coef, sizeof(double) * p);
  cudaStat5 = hipMalloc ((void**)&d_coef2, sizeof(double) * p);
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3); //check!!
  assert(hipSuccess == cudaStat4);
  assert(hipSuccess == cudaStat5);

  cudaStat1 = hipMemcpy(d_X, X, sizeof(double) * lda * p, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat1);
  cudaStat2 = hipMemcpy(d_Y, Y, sizeof(double) * n, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat2);
  double alpha_v = 1.0;
  double beta_v = 0.0;
  const double *alpha = &alpha_v, *beta = &beta_v; //check!!
  printf("%f\n", *alpha);
  // d_C = d_X^T d_X
  cublas_status = hipblasDgemm(cublasH,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           p, p, n, // DO NOT mess up the order
                           alpha,
                           d_X, n,
                           d_X, n,
                           beta,
                           d_C, p);
  cudaStat1 = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat1);
  printf("finish X'X\n");
  // copy d_C to C
  cudaStat1 = hipMemcpy(C, d_C, sizeof(double)*p*p, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);
  // inv(C)
  gsl_matrix *B = gsl_matrix_alloc(p, p);
  gsl_matrix_set_identity(B);

  cuda_LU_solve(C, p, B->data, p);
  cudaStat1 = hipMemcpy(d_C, B->data, sizeof(double)*p*p, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat1);
  for (int i = 0; i < p*p; i++)
    printf("%f\n", B->data[i]);
  gsl_matrix_free(B);
  printf("finish inv(C)\n");
  printf("%f %f\n", *alpha, *beta);
  // d_Y = d_X^T * d_Y
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           alpha,
                           d_X, n,
                           d_Y, 1,
                           beta,
                           d_coef, 1);
  cudaStat1 = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat1);
  cudaStat1 = hipMemcpy(coef, d_coef, sizeof(double) * p, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);
  for (int i = 0 ; i < p ; i ++ )
    printf("%f\n", coef[i]);

  // inv(C) * d_Y
  // due to by-column in gpu while by-row in gsl, C need to be transpose
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           p, p,
                           alpha,
                           d_C, p,
                           d_coef, 1,
                           beta,
                           d_coef2, 1);
  cudaStat1 = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat1);

  // copy to coef
  cudaStat1 = hipMemcpy(coef, d_coef2, sizeof(double) * p, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);
  for (int i = 0 ; i < p ; i ++ )
    printf("%f\n", coef[i]);
  hipFree(d_X);
  hipFree(d_Y);
  hipFree(d_C);

  hipblasDestroy(cublasH);
  hipDeviceReset();
  return 0;
}
